
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cassert>



__global__
void vectorAdd(int *d_a, int *d_b, int *d_c, int N) {
    int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    /* if the grid dimension (threads * blocks)
       is smaller than N, we can use a stride loop */

    for(int i=thread_id; i<N; i+=stride)
        d_c[thread_id] = d_a[thread_id] + d_b[thread_id];
}

void init(int *a, int *b, int N){
    for(int i=0; i<N; i++){
        a[i] = 10;
        b[i] = 10;
    }
}

void checkResult(int *a, int *b, int *c, int N){
    for(int i=0; i < N; i++)
        assert(c[i] = a[i] + b[i]);
    std::cout << "SUCCESS" <<std::endl;
}

int main(){
    int N = 1<<16; // Array size of 2^16 (65536 elts)
    size_t bytes = N * sizeof(int); // Needed memory space
    
    int *h_a, *h_b, *h_c;
    hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_c, bytes, hipHostMallocDefault);

    init(h_a, h_b, N);


    // create some storage to welcome the arrays on the device
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // (HtoD)
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    size_t threads = 1024;
    size_t blocks = (N + threads - 1) / threads;  // to ensure that we will have enough blocks (padding)
    
    vectorAdd<<<blocks, threads>>>(d_a, d_b, d_c, N);

    // (DtoH)
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    
    checkResult(h_a, h_b, h_c, N);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    return 0;
}